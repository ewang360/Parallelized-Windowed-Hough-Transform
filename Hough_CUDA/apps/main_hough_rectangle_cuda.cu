#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <iostream>
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <Eigen/Dense>
#include <cereal/archives/binary.hpp>
#include <cereal/archives/json.hpp>
#include <cereal/cereal.hpp>
#include <fstream>
#include "config.hpp"
#include "cxxopts.hpp"
#include "eigen_utils.hpp"
#include "io.hpp"
#include "process_image.hpp"
#include "rectangle_detection.hpp"
#include "rectangle_utils.hpp"
#include "stb_image.h"
#include "stb_image_write.h"
#include "string"
#include <fstream>
#include <chrono>
#include <vector>

// using Eigen::Dynamic;
// using Eigen::Matrix;
// using Eigen::RowMajor;
using namespace Eigen;
#define PI 3.14159265

std::chrono::high_resolution_clock::time_point startTime, endTime;

int main(int argc, char* argv[]) {

    // Nota bene: casting big images to unsigned char in Eigen result in a
    // segmentation fault on my machine for some unknown reasons. Compiler
    // complains that the array is too big. We have therefore chosen the
    // following way to convert Eigen matrix to unsigned char *

    int id = hipGetDevice(&id);
    std::cout << "cuda device id: " << id << std::endl;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Parse arguments
    cxxopts::Options options("Runs Hough rectangle detection algorithm");
    options.add_options()("i,image_path", "Path to binary (0-255) input image", cxxopts::value<std::string>())(
        "o,output_path", "Path to .txt file where detected rectangles will be saved", cxxopts::value<std::string>());
    auto result = options.parse(argc, argv);

    std::string filename = result["image_path"].as<std::string>();
    std::string output_filename = result["output_path"].as<std::string>();

    // Parse config file
    Config config;
    std::ifstream is("../src/configs.json");
    cereal::JSONInputArchive archive(is);
    archive(config);

    // Load image and prepare matrix
    Matrix<float, Dynamic, Dynamic, RowMajor> gray = eigen_io::read_image(filename.c_str());
    std::cout << "cuda version" << std::endl;

    std::vector<double> times;
    double avg_time = 0;

    //for (int t=0; t<10; t++) {    
    // get start time
    startTime = std::chrono::high_resolution_clock::now();

    // Perform Hough transform
    HoughRectangle ht(config.L_window, config.thetaBins, config.rhoBins, config.thetaMin, config.thetaMax);

    // Loop over each pixel to find rectangle
    rectangles_T<int> rectangles;
    HoughRectangle::fMat hough_img(config.rhoBins, config.thetaBins);

    double rho_min = -sqrt(pow(config.L_window / 2.0, 2) + pow(config.L_window / 2.0, 2));
    double rho_step = -rho_min*2/config.rhoBins;

    int found = 0;

    VectorXi vecX = VectorXi::LinSpaced(Sequential, config.L_window, 0, config.L_window - 1);
    VectorXi vecY = VectorXi::LinSpaced(Sequential, config.L_window, 0, config.L_window - 1);

    // Cartesian coordinate vectors
    int mid_X = round(config.L_window / 2);
    int mid_Y = round(config.L_window / 2);
    vecX = vecX.array() - mid_X;
    vecY = vecY.array() - mid_Y;

    int* d_vecX;
    float* d_cosT;
    int* d_vecY;
    float* d_sinT;
    float* d_img;
    float* d_acc;

    int m_theta_vec_size = config.thetaBins;
    int m_rho_vec_size = config.rhoBins;
    int size = config.L_window;
    size_t vec_X_size = sizeof(int) * size;
    size_t vec_Y_size = sizeof(int) * size;
    size_t img_size = sizeof(float) * size * size;
    size_t acc_size = sizeof(float) * m_theta_vec_size * m_rho_vec_size;
    size_t cosT_size = sizeof(float) * m_theta_vec_size;
    size_t sinT_size = sizeof(float) * m_theta_vec_size;

    hipMalloc((void **)&d_vecX, vec_X_size);
    hipMalloc((void **)&d_vecY, vec_Y_size);
    hipMalloc((void **)&d_cosT, cosT_size);
    hipMalloc((void **)&d_sinT, sinT_size);
    hipMalloc((void **)&d_img, img_size);
    hipMalloc((void **)&d_acc, acc_size);

    hipMemcpy(d_vecX, vecX.data(), vec_X_size, hipMemcpyHostToDevice);
    hipMemcpy(d_vecY, vecY.data(), vec_Y_size, hipMemcpyHostToDevice);

    for (int i = 0; i < gray.rows() - size; ++i) {
    // for (int i = 0; i < 1; ++i) {
        //std::cout << "Row " << i << "/" << gray.rows() << std::endl;
        for (int j = 0; j < gray.cols() - size; ++j) {
        // for (int j = 3; j < 4; ++j) {
            // if(j==0) {
            //     std::cout << "Row " << i << "/" << gray.rows() << std::endl;
            // }

            // Eigen::Vector3f *host_vectors = new Eigen::Vector3f[N];
            // Eigen::Vector3f *dev_vectors;

            hough_img.setZero();     
            hipMemset(d_acc, 0, acc_size);   
            ht.hough_transform(gray.block(i, j, config.L_window, config.L_window), hough_img, rho_min, rho_step, d_vecX, d_vecY, d_cosT, d_sinT, d_img, d_acc);
            // ht.hough_transform(gray.block(i, j, config.L_window, config.L_window), hough_img);

            // Detect peaks
            std::vector<std::array<int, 2>> indexes = find_local_maximum(hough_img, config.min_side_length);
            std::vector<float> rho_maxs, theta_maxs;
            std::tie(rho_maxs, theta_maxs) = ht.index_rho_theta(indexes);

            // Find pairs
            std::vector<std::array<float, 4>> pairs =
                rectangle_detect::find_pairs(rho_maxs, theta_maxs, config.T_rho, config.T_theta, config.T_l);
            if (pairs.size() == 0) {
                continue;
            }  // no pairs detected

            // Find rectangle
            rectangles_T<float> rectangles_tmp = rectangle_detect::match_pairs_into_rectangle(pairs, config.T_alpha);
            if (rectangles_tmp.size() == 0) {
                continue;
            }  // if no rectangle detected
            // else {
            //     found = 1;
            //     // std::cout << "Rectangle detected"
            //     //         << " " << i << " " << j << std::endl;
            // }
            std::array<float, 8> detected_rectangle = rectangle_detect::remove_duplicates(rectangles_tmp, 1, 4);
            auto rectangles_corners =
                convert_all_rects_2_corner_format(detected_rectangle, config.L_window, config.L_window);
            correct_offset_rectangle(rectangles_corners, j, i);

            // Concatenate
            rectangles.push_back(rectangles_corners);
        }
    }
    
    // Free the allocated memory on the GPU
    hipFree(d_vecX);
    hipFree(d_cosT);
    hipFree(d_vecY);
    hipFree(d_sinT);
    hipFree(d_img);
    hipFree(d_acc);

    endTime = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = endTime - startTime;
    double elapsedTimeInSeconds = elapsed.count();

    times.push_back(elapsedTimeInSeconds);
    avg_time += elapsedTimeInSeconds;
    //}

    for (int i=0; i<1; i++) {
        std::cout << "Time elapsed: " << times[i] << " sec" << std::endl;
    }

    // std::cout << "Average time elapsed: " << avg_time/10 << std::endl;

    // if (found == 0) {
    //     std::cout << "Did not detect any rectangle" << std::endl;
    //     exit(0);
    // }

    // Clean up and save
    eigen_io::save_rectangle(output_filename.c_str(), rectangles);

    return 0;
}
