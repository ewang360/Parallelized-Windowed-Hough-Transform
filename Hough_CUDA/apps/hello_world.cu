#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <Eigen/Dense>
#include <cereal/archives/binary.hpp>
#include <cereal/archives/json.hpp>
#include <cereal/cereal.hpp>
#include <fstream>
#include "config.hpp"
#include "cxxopts.hpp"
#include "eigen_utils.hpp"
#include "io.hpp"
#include "process_image.hpp"
#include "rectangle_detection.hpp"
#include "rectangle_utils.hpp"
#include "stb_image.h"
#include "stb_image_write.h"
#include "string"
#include <fstream>
#include <chrono>
#include <vector>

__global__ void hello( int* n ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x ; // Which one am
    // if(n[i] <= 15) {
        printf( "Hello world from thread %d!\n", i ); // What do I
    // }
    // else {
    //     printf( "Goodbye world from thread %d!\n", i );
    // }
}

int main( void ) {
    int *ptr = new int[32];

    for(int i=0; i<32; i++) {
        ptr[i] = i;
    }
    // printf( "Running Kernel A \n" );
    // hello<<< 1,1 >>>( 1 );
    // hipDeviceSynchronize();

    printf( "Running Kernel B \n" );
    hello<<< 1,32 >>>( ptr );
    hipDeviceSynchronize();

    // printf( "Running Kernel C \n" );
    // hello<<< 8,32 >>>( 1 );
    // hipDeviceSynchronize();

    return 0;
}